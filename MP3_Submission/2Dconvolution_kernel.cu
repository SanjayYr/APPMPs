#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix N, Matrix P)
{
   __shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];

   int bx = blockIdx.x;    int tx = threadIdx.x;
   int by = blockIdx.y;    int ty = threadIdx.y;

   int row_o = by * TILE_SIZE + ty;
   int col_o = bx * TILE_SIZE + tx;


   int row_i = row_o - KS_DIV_2;
   int col_i = col_o - KS_DIV_2;

   if((row_i >= 0) && (row_i < N.height) &&
       (col_i >= 0) && (col_i < N.width))
   {
      N_s[ty][tx] = N.elements[row_i * N.width + col_i];
   }
   else
   {
      N_s[ty][tx] = 0.0f;
   }
   __syncthreads();

   float Pvalue = 0.0f;
   if(ty < TILE_SIZE && tx < TILE_SIZE) 
   {
      for(int i=0; i < KERNEL_SIZE; i++)
      {
         for(int j=0; j < KERNEL_SIZE; j++)
         {
            Pvalue += Mc[i*KERNEL_SIZE + j] * N_s[i + ty][j + tx];
         }
      }
      __syncthreads();
      if(row_o < P.height && col_o < P.width)
      {
         P.elements[row_o * P.width + col_o] = Pvalue;
      }
   }
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
